
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

void cpu(int *a,int N){
    for(int i=0;i<N;i++){
        a[i]=i;
    }
}

__global__ void gpu(int *a,int N){
    int threadi=blockIdx.x*blockDim.x+threadIdx.x;
    int stride=gridDim.x*blockDim.x;
    for(int i=threadi;i<N;i+=stride){
        a[i]*=2;
    }
}

bool check(int *a,int N){
    for(int i=0;i<N;i++){
        if(a[i]!=2*i){
            
        return false;
        }
    }
    return true;
}

int main(){
    const int N=1000;
    size_t size=N*sizeof(int);

    int *a;
    hipMallocManaged(&a,size);

    // 运行cpu
    cpu(a,N);

    //运行gpu
    size_t threads=256;
    size_t blocks=1;
    gpu<<<blocks,threads>>>(a,N);
    hipDeviceSynchronize();

    //运行check函数
    check(a,N)?printf("OK"):printf("Error");
    hipFree(a);
}
