
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<assert.h>

void cpu(int *a,int N){
    for(int i=0;i<N;i++){
        a[i]=i;
    }
}

__global__ void gpu(int *a,int N){
    int threadi=blockIdx.x*blockDim.x+threadIdx.x;
    int stride=gridDim.x*blockDim.x;
    for(int i=threadi;i<N;i+=stride){
        a[i]*=2;
    }
}

bool check(int *a,int N){
    for(int i=0;i<N;i++){
        if(a[i]!=2*i){
            
        return false;
        }
    }
    return true;
}

// 错误处理宏
inline hipError_t checkCuda(hipError_t result){
    if(result!=hipSuccess)
    {
        fprintf(stderr,"CUDA runtime error:%s\n",hipGetErrorString(result));
        assert(result==hipSuccess);
    }
    return result;
}

int main(){
    const int N=1000;
    size_t size=N*sizeof(int);

    int *a;
    hipError_t error;
    error=hipMallocManaged(&a,size);
    if(error!=hipSuccess){
        printf("Error:%s\n",hipGetErrorString(error));
    }

    // 运行cpu
    cpu(a,N);

    //运行gpu
    size_t threads=256;
    size_t blocks=1;
    gpu<<<blocks,-1>>>(a,N);

    error=hipGetLastError();
    if(error!=hipSuccess){
        printf("Error:%s\n",hipGetErrorString(error));
    }

    checkCuda(hipDeviceSynchronize());

    //运行check函数
    check(a,N)?printf("OK"):printf("Error");
    hipFree(a);
}
