
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

void cpu(int *a,int N){
    for(int i=0;i<N;i++){
        a[i]=i;
    }
}

__global__ void gpu(int *a,int N){
    int threadi=blockIdx.x*blockDim.x+threadIdx.x;
    int stride=gridDim.x*blockDim.x;
    for(int i=threadi;i<N;i+=stride){
        a[i]*=2;
    }
}

bool check(int *a,int N){
    for(int i=0;i<N;i++){
        if(a[i]!=2*i){
            
        return false;
        }
    }
    return true;
}

int main(){
    const int N=1000;
    size_t size=N*sizeof(int);

    int *a;
    hipError_t error;
    error=hipMallocManaged(&a,size);
    if(error!=hipSuccess){
        printf("Error:%s\n",hipGetErrorString(error));
    }

    // 运行cpu
    cpu(a,N);

    //运行gpu
    size_t threads=256;
    size_t blocks=1;
    gpu<<<blocks,-1>>>(a,N);

    error=hipGetLastError();
    if(error!=hipSuccess){
        printf("Error:%s\n",hipGetErrorString(error));
    }

    hipDeviceSynchronize();

    //运行check函数
    check(a,N)?printf("OK"):printf("Error");
    hipFree(a);
}
