
#include <hip/hip_runtime.h>
#include<stdio.h>

int main(){
    int id;
    hipGetDevice(&id);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props,id);

    printf("device id: %d\n sms: %d\n capability major :%d\n capability minor :%d\n wrap size : %d\n",id,props.multiProcessorCount,props.major,props.minor,props.warpSize);

}
