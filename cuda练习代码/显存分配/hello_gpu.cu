
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

void cpu(int *a,int N){
    for(int i=0;i<N;i++){
        a[i]=i;
    }
}

__global__ void gpu(int *a,int N){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if(i<N){
        a[i]*=2;
    }
}

bool check(int *a,int N){
    for(int i=0;i<N;i++){
        if(a[i]!=2*i)
        return false;
    }
    return true;
}

int main(){
    const int N=1000;
    size_t size=N*sizeof(int);

    int *a;
    hipMallocManaged(&a,size);

    // 运行cpu
    cpu(a,N);

    //运行gpu
    size_t threads=256;
    size_t blocks=(N+threads-1)/threads;
    gpu<<<blocks,threads>>>(a,N);
    hipDeviceSynchronize();

    //运行check函数
    check(a,N)?printf("OK"):printf("Error");
    hipDeviceSynchronize();
}
